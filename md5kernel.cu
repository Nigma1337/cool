#include "hip/hip_runtime.h"
//This is our CUDA thread
//d_a is the word list array
//maxidx is the maximum index in the array (if there are more threads than words)
#include <stdint.h>

#define N_PERFECT_MATCH 11
#define N_NICE_MATCH 10
#define N_GOLD_MD5 7
#define N_MD5_OF_DIGITS 32
#define N_MD5_OF_LETTERS 23
#define N_PI_MD5 9
#define N_E_MD5 9

typedef struct {
    uint8_t data[16];
} md5_digest_t;

__device__ bool is_perfect_match(char *hash) {
    for (int i = 1; i < 32; ++i) {
        if (hash[i] != hash[0]) {
            return false;
        }
    }
    return true;
}

__device__ bool is_nice_match(char *hash) {
    for (int i = 1; i < N_NICE_MATCH; ++i) {
        if (hash[i] != hash[0]) {
            return false;
        }
    }
    return true;
}

__device__ bool is_gold_md5(char *text, char *hash) {
    for (int i = 0; i < N_GOLD_MD5; ++i) {
        if (text[i] != hash[i]) {
            return false;
        }
    }
    return true;
}

__device__ bool is_pi_md5(char *hash) {
    char pi_str[35];
    for (int i = 0; i < N_PI_MD5; ++i) {
        if (hash[i] != pi_str[i + 2]) {
            return false;
        }
    }
    return true;
}

__device__ bool is_e_md5(char *hash) {
    char e_str[35];
    for (int i = 0; i < N_E_MD5; ++i) {
        if (hash[i] != e_str[i + 2]) {
            return false;
        }
    }
    return true;
}

__device__ size_t check_nice_match(md5_digest_t digest) {
    uint8_t first = digest.data[0];
    if (first >> 4 != first & 0xF) {
        return 0;
    }

    uint8_t expected = first & 0xF;
    for (int i = 1; i < 16; i++) {
        uint8_t byte = digest.data[i];
        if (byte >> 4 != expected) {
            return 2 * i;
        }
        if (byte & 0xF != expected) {
            return 2 * i + 1;
        }
    }

    return 32;
}

__device__ void IncrementBruteGPU(unsigned char* ourBrute, uint charSetLen, uint bruteLength, uint incrementBy)
{
	int i = 0;
	while(incrementBy > 0 && i < bruteLength)
	{
		int add = incrementBy + ourBrute[i];
		ourBrute[i] = add % charSetLen;
		incrementBy = add / charSetLen;
		i++;
	}
}

__global__ void crack(uint numThreads, uint charSetLen, uint bruteLength)
{
	//compute our index number
    	uint idx = (blockIdx.x*blockDim.x + threadIdx.x);
	int totalLen = 0;
	int bruteStart = 0;

	unsigned char word[MAX_TOTAL];
	unsigned char ourBrute[MAX_BRUTE_LENGTH];
	int i = 0;

	for(i = 0; i < MAX_BRUTE_LENGTH; i++)
	{
		ourBrute[i] = cudaBrute[i];
	}
	
	IncrementBruteGPU(ourBrute, charSetLen, bruteLength, idx);
	int timer = 0;
	for(timer = 0; timer < MD5_PER_KERNEL; timer++)
	{	
		//Now, substitute the values into the string
		for(i = 0; i < bruteLength; i++)
		{
			word[i+bruteStart] = cudaCharSet[ourBrute[i]];
		}

		uint c1 = 0, c2 = 0, c3 = 0, c4 = 0;
		//get the md5 hash of the word
		md5_vfy(word,totalLen, &c1, &c2, &c3, &c4);
		IncrementBruteGPU(ourBrute, charSetLen, bruteLength, numThreads);
	}
}